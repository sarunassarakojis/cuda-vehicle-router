#include "hip/hip_runtime.h"
#include "cuda_router_library.cuh"
#include <stdio.h>

__global__
void cuda_vr::execute_kernel() {
	printf("Thread %d running%c", threadIdx.x, '\n');
}

void cuda_vr::execute_kernel_wrapper(int num_of_threads) {
	execute_kernel<<<1, num_of_threads>>>();
}

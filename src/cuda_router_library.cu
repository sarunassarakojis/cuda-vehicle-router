#include "hip/hip_runtime.h"
#include "cuda_router_library.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

__global__
void add_vectors(float* source_vec_a, float* source_vec_b, float* res_vec, int size) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        res_vec[i] = source_vec_a[i] + source_vec_b[i];
    }
}

template <typename T>
void print_array_contents(T* block, size_t size) {
    for (auto i = 0; i != size; ++i) {
        std::cout << "Element: " << block[i] << std::endl;
    }
}

void cuda_vr::add_vectors_wrapper() {
    const int n = 5;
    size_t size = n * sizeof(float);
    float* vec_a_host = (float*)malloc(size);
    float* vec_b_host = (float*)malloc(size);
    float* vec_res_host = (float*)malloc(size);
    float* vec_a_device;
    float* vec_b_device;
    float* vec_res_device;

    vec_a_host[0] = 2.3f;
    vec_a_host[1] = 2.5f;
    vec_a_host[2] = 1.0f;
    vec_a_host[3] = 999.1f;
    vec_a_host[4] = .1f;
    std::cout << "Source vector a:" << std::endl;
    print_array_contents(vec_a_host, n);
    std::cout << std::endl;

    vec_b_host[0] = 2.3f;
    vec_b_host[1] = 2.5f;
    vec_b_host[2] = 1.0f;
    vec_b_host[3] = 999.1f;
    vec_b_host[4] = .1f;

    hipMalloc(&vec_a_device, size);
    hipMalloc(&vec_b_device, size);
    hipMalloc(&vec_res_device, size);

    hipMemcpy(vec_a_device, vec_a_host, size, hipMemcpyHostToDevice);
    hipMemcpy(vec_b_device, vec_b_host, size, hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

    add_vectors<<<blocks_per_grid, threads_per_block>>>(vec_a_device, vec_b_device, vec_res_device, n);

    hipMemcpy(vec_res_host, vec_res_device, size, hipMemcpyDeviceToHost);

    std::cout << "Result vector:" << std::endl;
    print_array_contents(vec_res_host, n);

    hipFree(vec_a_device);
    hipFree(vec_b_device);
    hipFree(vec_res_device);
}

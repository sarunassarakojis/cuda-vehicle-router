#include "hip/hip_runtime.h"
#include "router_libraries.cuh"
#include <stdio.h>

__global__
void vr::execute_kernel() {
	printf("Thread %d running%c", threadIdx.x, '\n');
}

void vr::execute_kernel_wrapper(int num_threads) {
	vr::execute_kernel<<<1, num_threads>>>();
}

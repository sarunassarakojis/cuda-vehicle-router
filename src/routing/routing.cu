#include "hip/hip_runtime.h"
#include "routing/routing.h"

#include <iostream>
#include <algorithm>
#include <forward_list>
#include <unordered_set>

using namespace std;
using namespace routing;

inline double power_by_2(const double& x) {
    return pow(x, 2);
}

inline double** get_distances_matrix(vector<Node>& nodes) {
    const int n = nodes.size();
    double** distance_matrix = new double*[n];

    for (int i = 0; i < n; ++i) {
        distance_matrix[i] = new double[n];

        for (int j = 0; j < n; j++) {
            double d = sqrt(power_by_2(nodes[i].x - nodes[j].x) + power_by_2(nodes[i].y - nodes[j].y));

            distance_matrix[i][j] = d;
        }
    }

    return distance_matrix;
}

inline void calculate_savings(vector<Saving>& savings, double** distance_matrix, const int& size) {
    for (int i = 1; i < size; ++i) {
        for (int j = i + 1; j < size; ++j) {
            savings.push_back(Saving{
                i, j, distance_matrix[0][i] + distance_matrix[0][j] - distance_matrix[i][j]
            });
        }
    }
}

std::forward_list<Route> routing::route(vector<Node> nodes, unsigned vehicle_capacity) {
    const int n = nodes.size();
    double** distance_matrix = get_distances_matrix(nodes);
    vector<Saving> savings;
    forward_list<Route> routes;
    unordered_set<long> added_nodes_index;

    added_nodes_index.reserve(n);
    savings.reserve(power_by_2(n));

    calculate_savings(savings, distance_matrix, n);
    sort(savings.begin(), savings.end(), [&](auto& s1, auto& s2) -> bool { return s1.saving > s2.saving; });

    // main algo
    for (size_t i = 0, savings_n = savings.size(); added_nodes_index.size() != n && i != savings_n; i++) {
        const auto saving = savings[i];
        const auto found_i = added_nodes_index.find(saving.node_i) != added_nodes_index.end();
        const auto found_j = added_nodes_index.find(saving.node_j) != added_nodes_index.end();

        if (!found_i && !found_j && nodes[saving.node_i].demand + nodes[saving.node_j].demand <= vehicle_capacity) {

            added_nodes_index.insert(saving.node_i);
            added_nodes_index.insert(saving.node_j);
            routes.push_front(Route{
                nodes[saving.node_i].demand + nodes[saving.node_j].demand,
                {saving.node_i, saving.node_j}
            });
        }
        else if (found_i && !found_j) {
            // TODO refactor
            for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
                if (iterator->nodes.front() == saving.node_i
                    && iterator->met_demand + saving.node_j <= vehicle_capacity) {
                    iterator->met_demand += nodes[saving.node_j].demand;
                    iterator->nodes.push_front(saving.node_j);
                    added_nodes_index.insert(saving.node_j);
                    break;
                }
                if (iterator->nodes.back() == saving.node_i
                    && iterator->met_demand + saving.node_j <= vehicle_capacity) {
                    iterator->met_demand += nodes[saving.node_j].demand;
                    iterator->nodes.push_back(saving.node_j);
                    added_nodes_index.insert(saving.node_j);
                    break;
                }
            }
        }
        else if (found_j && !found_i) {
            // TODO refactor
            for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
                if (iterator->nodes.front() == saving.node_j
                    && iterator->met_demand + saving.node_i <= vehicle_capacity) {
                    iterator->met_demand += nodes[saving.node_i].demand;
                    iterator->nodes.push_front(saving.node_i);
                    added_nodes_index.insert(saving.node_i);
                    break;
                }
                if (iterator->nodes.back() == saving.node_j
                    && iterator->met_demand + saving.node_i <= vehicle_capacity) {
                    iterator->met_demand += nodes[saving.node_i].demand;
                    iterator->nodes.push_back(saving.node_i);
                    added_nodes_index.insert(saving.node_i);
                    break;
                }
            }
        }
    }

    if (added_nodes_index.size() != n) {
        auto end = added_nodes_index.end();

        for (auto i = 1; i < n; ++i) {
            if (added_nodes_index.find(nodes[i].indice) == end) {
                routes.push_front(Route{nodes[i].demand, {nodes[i].indice}});
            }
        }
    }

    for (auto i = 0; i < n; i++) {
        delete[] distance_matrix[i];
    }
    delete[] distance_matrix;

    return routes;
}

std::forward_list<routing::Route> routing::route_parallel(std::vector<Node> nodes, unsigned vehicle_capacity) {
    return forward_list<Route>{};
}

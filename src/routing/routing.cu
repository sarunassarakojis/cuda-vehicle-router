#include "hip/hip_runtime.h"
#include "routing/routing.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <forward_list>
#include <unordered_set>

inline double power_by_2(const double& x) {
    return pow(x, 2);
}

template <typename T>
void print_out(T** matrix, const int& size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            std::cout << matrix[i][j] << " ";
        }
        std::cout << "\n";
    }
}

std::forward_list<routing::Route> routing::route(std::vector<Node> nodes, unsigned vehicle_capacity) {
    using namespace std;

    const int n = nodes.size();
    double** distance_matrix = new double*[n];
    vector<Saving> savings;
    forward_list<Route> routes;
    unordered_set<long> added_nodes_index;

    added_nodes_index.reserve(n);

    // distance matrix calculation
    for (int i = 0; i < n; i++) {
        distance_matrix[i] = new double[n];

        for (int j = 0; j < n; j++) {
            double d = sqrt(power_by_2(nodes[i].x - nodes[j].x) + power_by_2(nodes[i].y - nodes[j].y));

            distance_matrix[i][j] = d;
        }
    }

    cout << "Distance matrix:" << '\n';
    print_out(distance_matrix, n);

    // savings calculation
    savings.reserve(power_by_2(n));
    for (int i = 1; i < n; i++) {
        for (int j = 1; j < n; j++) {
            if (i != j) {
                double saving = distance_matrix[0][i] + distance_matrix[0][j] - distance_matrix[i][j];

                savings.push_back(Saving{i, j, saving});
            }
        }
    }

    // sort saving in descending order
    sort(savings.begin(), savings.end(), [&](auto& s1, auto& s2) -> bool { return s1.saving > s2.saving; });

    cout << "\nSavings sorted:" << '\n';
    for (auto& s : savings) {
        printf("s(%d, %d) = %f\n", s.node_i, s.node_j, s.saving);
    }

    // main algo
    for (auto& saving : savings) {
        const auto found_i = added_nodes_index.find(saving.node_i) != added_nodes_index.end();
        const auto found_j = added_nodes_index.find(saving.node_j) != added_nodes_index.end();

        if (!found_i && !found_j && nodes[saving.node_i].demand + nodes[saving.node_j].demand <= vehicle_capacity) {

            added_nodes_index.insert(saving.node_i);
            added_nodes_index.insert(saving.node_j);
            routes.push_front(Route{
                nodes[saving.node_i].demand + nodes[saving.node_j].demand,
                {saving.node_i, saving.node_j}
            });
        }
        else if (found_i && !found_j) {
            // TODO refactor
            // TODO also check if constraints are not violated
            // TODO update route_cost as well
            for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
                if (iterator->nodes.front() == saving.node_i) {
                    iterator->nodes.push_front(saving.node_j);
                    added_nodes_index.insert(saving.node_j);
                    break;
                }
                if (iterator->nodes.back() == saving.node_i) {
                    iterator->nodes.push_back(saving.node_j);
                    added_nodes_index.insert(saving.node_j);
                    break;
                }
            }
        }
        else if (found_j && !found_i) {
            // TODO refactor
            // TODO also check if constraints are not violated
            // TODO update route_cost as well
            for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
                if (iterator->nodes.front() == saving.node_j) {
                    iterator->nodes.push_front(saving.node_i);
                    added_nodes_index.insert(saving.node_i);
                    break;
                }
                if (iterator->nodes.back() == saving.node_j) {
                    iterator->nodes.push_back(saving.node_i);
                    added_nodes_index.insert(saving.node_i);
                    break;
                }
            }
        }

    }

    for (auto i = 0; i < n; i++) {
        delete[] distance_matrix[i];
    }
    delete[] distance_matrix;

    return routes;
}

#include "hip/hip_runtime.h"
#include "routing/routing.h"

#include <iostream>
#include <algorithm>
#include <forward_list>
#include <unordered_set>

using namespace std;
using namespace routing;

inline double power_by_2(const double& x) {
    return pow(x, 2);
}

inline double** get_distances_matrix(vector<Node>& nodes) {
    const int n = nodes.size();
    double** distance_matrix = new double*[n];

    for (int i = 0; i < n; ++i) {
        distance_matrix[i] = new double[n];

        for (int j = 0; j < n; j++) {
            double d = sqrt(power_by_2(nodes[i].x - nodes[j].x) + power_by_2(nodes[i].y - nodes[j].y));

            distance_matrix[i][j] = d;
        }
    }

    return distance_matrix;
}

inline void calculate_savings(vector<Saving>& savings, double** distance_matrix, const int& size) {
    for (int i = 1; i < size; ++i) {
        for (int j = i + 1; j < size; ++j) {
            savings.push_back(Saving{
                i, j, distance_matrix[0][i] + distance_matrix[0][j] - distance_matrix[i][j]
            });
        }
    }
}

inline void append_new_node_to_route(forward_list<Route>& routes, long& found_node, long& new_node,
                                     unsigned& capacity, unsigned& node_demand, unordered_set<long>& added_nodes) {

    for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
        if (iterator->nodes.front() == found_node && iterator->met_demand + node_demand <= capacity) {
            iterator->met_demand += node_demand;
            iterator->nodes.push_front(new_node);
            added_nodes.insert(new_node);
            break;
        }
        if (iterator->nodes.back() == found_node && iterator->met_demand + node_demand <= capacity) {
            iterator->met_demand += node_demand;
            iterator->nodes.push_back(new_node);
            added_nodes.insert(new_node);
            break;
        }
    }
}

inline void add_unoptimized_routes(unordered_set<long>& added_nodes, forward_list<Route>& routes,
    vector<Node>& nodes, const int& size, const unsigned& capacity) {
    auto end = added_nodes.end();

    for (auto i = 1; i < size; ++i) {
        if (added_nodes.find(nodes[i].indice) == end
            && nodes[i].demand <= capacity) {
            routes.push_front(Route{ nodes[i].demand,{ nodes[i].indice } });
        }
    }
}

std::forward_list<Route> routing::route(vector<Node> nodes, unsigned vehicle_capacity) {
    const int n = nodes.size();
    double** distance_matrix = get_distances_matrix(nodes);
    vector<Saving> savings;
    forward_list<Route> routes;
    unordered_set<long> added_nodes;

    added_nodes.reserve(n);
    savings.reserve(power_by_2(n));

    calculate_savings(savings, distance_matrix, n);
    sort(savings.begin(), savings.end(), [&](auto& s1, auto& s2) -> bool { return s1.saving > s2.saving; });

    for (size_t i = 0, savings_n = savings.size(); added_nodes.size() != n && i != savings_n; i++) {
        const auto saving = savings[i];
        auto node_i = saving.node_i;
        auto node_j = saving.node_j;
        auto node_i_demand = nodes[node_i].demand;
        auto node_j_demand = nodes[node_j].demand;
        const auto found_i = added_nodes.find(node_i) != added_nodes.end();
        const auto found_j = added_nodes.find(node_j) != added_nodes.end();

        if (!found_i && !found_j && node_i_demand + node_j_demand <= vehicle_capacity) {
            added_nodes.insert(node_i);
            added_nodes.insert(node_j);
            routes.push_front(Route{node_i_demand + node_j_demand, {node_i, node_j}});
        }
        else if (found_i && !found_j) {
            append_new_node_to_route(routes, node_i, node_j, vehicle_capacity, node_j_demand, added_nodes);
        }
        else if (found_j && !found_i) {
            append_new_node_to_route(routes, node_j, node_i, vehicle_capacity, node_i_demand, added_nodes);
        }
    }

    if (added_nodes.size() != n) {
        add_unoptimized_routes(added_nodes, routes, nodes, n, vehicle_capacity);
    }

    for (auto i = 0; i < n; i++) {
        delete[] distance_matrix[i];
    }
    delete[] distance_matrix;

    return routes;
}

forward_list<Route> routing::route_parallel(vector<Node> nodes, unsigned vehicle_capacity) {
    return forward_list<Route>{};
}

#include "hip/hip_runtime.h"
#include "routing/routing.h"
#include "utilities/logging.h"

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <forward_list>
#include <unordered_set>

using namespace std;
using namespace routing;
using namespace logging;

struct Savings_ordering_desc {

    __host__ __device__ bool operator()(Saving& s1, Saving& s2) const {
        return s1.saving > s2.saving;
    }
};

inline float power_by_2(const float& x) {
    return powf(x, 2);
}

inline float** get_distances_matrix(vector<Node>& nodes) {
    const int n = nodes.size();
    float** distance_matrix = new float*[n];

    for (int i = 0; i < n; ++i) {
        distance_matrix[i] = new float[n];

        for (int j = 0; j < n; j++) {
            auto d = sqrtf(power_by_2(nodes[i].x - nodes[j].x) + power_by_2(nodes[i].y - nodes[j].y));

            distance_matrix[i][j] = d;
        }
    }

    return distance_matrix;
}

inline void calculate_savings(vector<Saving>& savings, float** distance_matrix, const int& size) {
    for (int i = 1; i < size; ++i) {
        for (int j = i + 1; j < size; ++j) {
            savings.push_back(Saving{
                i, j, distance_matrix[0][i] + distance_matrix[0][j] - distance_matrix[i][j]
            });
        }
    }
}

inline void append_new_node_to_route(forward_list<Route>& routes, long& found_node, long& new_node,
                                     unsigned& capacity, unsigned& node_demand, unordered_set<long>& added_nodes) {

    for (auto iterator = routes.begin(); iterator != routes.end(); ++iterator) {
        if (iterator->nodes.front() == found_node && iterator->met_demand + node_demand <= capacity) {
            iterator->met_demand += node_demand;
            iterator->nodes.push_front(new_node);
            added_nodes.insert(new_node);
            break;
        }
        if (iterator->nodes.back() == found_node && iterator->met_demand + node_demand <= capacity) {
            iterator->met_demand += node_demand;
            iterator->nodes.push_back(new_node);
            added_nodes.insert(new_node);
            break;
        }
    }
}

inline void add_unoptimized_routes(unordered_set<long>& added_nodes, forward_list<Route>& routes,
                                   Node* nodes, int& size, unsigned& capacity) {
    auto end = added_nodes.end();

    for (auto i = 1; i < size; ++i) {
        if (added_nodes.find(nodes[i].indice) == end
            && nodes[i].demand <= capacity) {
            routes.push_front(Route{nodes[i].demand, {nodes[i].indice}});
        }
    }
}

std::forward_list<Route> routing::route(vector<Node> nodes, unsigned vehicle_capacity) {
    int size = nodes.size();
    float** distance_matrix = get_distances_matrix(nodes);
    vector<Saving> savings;
    forward_list<Route> routes;
    unordered_set<long> added_nodes;

    added_nodes.reserve(size);
    savings.reserve(power_by_2(size));

    calculate_savings(savings, distance_matrix, size);
    sort(savings.begin(), savings.end(), [&](auto& s1, auto& s2) -> bool { return s1.saving > s2.saving; });

    for (size_t i = 0, savings_n = savings.size(); added_nodes.size() != size - 1 && i != savings_n; i++) {
        const auto saving = savings[i];
        auto node_i = saving.node_i;
        auto node_j = saving.node_j;
        auto node_i_demand = nodes[node_i].demand;
        auto node_j_demand = nodes[node_j].demand;
        const auto found_i = added_nodes.find(node_i) != added_nodes.end();
        const auto found_j = added_nodes.find(node_j) != added_nodes.end();

        if (!found_i && !found_j && node_i_demand + node_j_demand <= vehicle_capacity) {
            added_nodes.insert(node_i);
            added_nodes.insert(node_j);
            routes.push_front(Route{node_i_demand + node_j_demand, {node_i, node_j}});
        }
        else if (found_i && !found_j) {
            append_new_node_to_route(routes, node_i, node_j, vehicle_capacity, node_j_demand, added_nodes);
        }
        else if (found_j && !found_i) {
            append_new_node_to_route(routes, node_j, node_i, vehicle_capacity, node_i_demand, added_nodes);
        }
    }

    if (added_nodes.size() != size - 1) {
        add_unoptimized_routes(added_nodes, routes, &nodes[0], size, vehicle_capacity);
    }

    for (auto i = 0; i < size; i++) {
        delete[] distance_matrix[i];
    }
    delete[] distance_matrix;

    return routes;
}

inline void log_cuda_error(hipError_t error, char* message = "CUDA error: {}") {
    get_logger()->error(message, hipGetErrorString(error));
}

__global__ void calculate_distance_matrix(Node* nodes, float* distance_matrix, int size) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < size && y < size) {
        distance_matrix[size * y + x] = sqrtf(
            powf(nodes[x].x - nodes[y].x, 2) + powf(nodes[x].y - nodes[y].y, 2));
    }
}

__global__ void calculate_savings(float* distance_matrix, Saving* savings, int size) {
    int y = blockDim.x * blockIdx.x + threadIdx.x;
    int x = blockDim.y * blockIdx.y + threadIdx.y;

    if (y > 0 && x > 0 && x < size && y < size) {
        int array_index = (size - 1) * (y - 1) + (x - 1);
        Saving& saving = savings[array_index];

        saving.node_i = x;
        saving.node_j = y;
        saving.saving = x != y ? distance_matrix[y] + distance_matrix[x] - distance_matrix[size * x + y] : 0;
    }
}

inline void sort_savings_desc(Saving* savings, int& savings_size) {
    thrust::sort(thrust::device_ptr<Saving>(savings),
                 thrust::device_ptr<Saving>(savings + savings_size), Savings_ordering_desc());
}

forward_list<Route> routing::route_parallel(Node* nodes, int size, unsigned vehicle_capacity,
                                            Thread_config configuration) {
    auto savings_size = (size - 1) * (size - 1);
    float* distance_matrix_d;
    Node* nodes_d;
    Saving* savings_h = new Saving[savings_size];
    Saving* savings_d;
    forward_list<Route> routes;
    unordered_set<long> added_nodes;

    hipError_t error = hipMalloc((void**)&distance_matrix_d, size * size * sizeof(float));

    if (error != hipSuccess) {
        log_cuda_error(error);
        goto cleanup;
    }

    error = hipMalloc((void**)&nodes_d, size * sizeof(Node));

    if (error != hipSuccess) {
        log_cuda_error(error);
        goto cleanup;
    }

    error = hipMalloc((void**)&savings_d, savings_size * sizeof(Saving));

    if (error != hipSuccess) {
        log_cuda_error(error);
        goto cleanup;
    }

    error = hipMemcpy(nodes_d, nodes, size * sizeof(Node), hipMemcpyHostToDevice);

    if (error != hipSuccess) {
        log_cuda_error(error);
        goto cleanup;
    }

    dim3 threads_per_block(configuration.threads_per_block_x, configuration.threads_per_block_y);
    dim3 block_dim(size / threads_per_block.x + 1, size / threads_per_block.y + 1);

    calculate_distance_matrix<<<block_dim, threads_per_block>>>(nodes_d, distance_matrix_d, size);
    calculate_savings<<<block_dim, threads_per_block>>>(distance_matrix_d, savings_d, size);
    sort_savings_desc(savings_d, savings_size);

    error = hipMemcpy(savings_h, savings_d, savings_size * sizeof(Saving), hipMemcpyDeviceToHost);

    if (error != hipSuccess) {
        log_cuda_error(error);
        goto cleanup;
    }

    for (auto i = 0; i < savings_size && added_nodes.size() != size - 1; ++i) {
        const auto saving = savings_h[i];
        auto node_i = saving.node_i;
        auto node_j = saving.node_j;
        auto node_i_demand = nodes[node_i].demand;
        auto node_j_demand = nodes[node_j].demand;
        const auto found_i = added_nodes.find(node_i) != added_nodes.end();
        const auto found_j = added_nodes.find(node_j) != added_nodes.end();

        if (node_i != node_j && !found_i && !found_j && node_i_demand + node_j_demand <= vehicle_capacity) {
            added_nodes.insert(node_i);
            added_nodes.insert(node_j);
            routes.push_front(Route{node_i_demand + node_j_demand, {node_i, node_j}});
        }
        else if (found_i && !found_j) {
            append_new_node_to_route(routes, node_i, node_j, vehicle_capacity, node_j_demand, added_nodes);
        }
        else if (found_j && !found_i) {
            append_new_node_to_route(routes, node_j, node_i, vehicle_capacity, node_i_demand, added_nodes);
        }
    }

    if (added_nodes.size() != size - 1) {
        add_unoptimized_routes(added_nodes, routes, nodes, size, vehicle_capacity);
    }

cleanup:
    delete[] savings_h;
    hipFree(nodes_d);
    hipFree(distance_matrix_d);
    hipFree(savings_d);

    return routes;
}
